/*
 * NCCL AllGather Example on a Single Node with 4 GPUs
 *
 * Description:
 * This program demonstrates how to use NVIDIA's NCCL library in conjunction with CUDA
 * to perform an AllGather operation across 4 GPUs on a single node.
 * Each GPU starts with a distinct vector (with one non-zero value),
 * and after the AllGather, all GPUs receive the combined vector [10, 20, 30, 40].
 *
 * Components:
 *  - CUDA for device memory allocation, kernel launch, and synchronization
 *  - NCCL for collective communication (AllGather)
 *  - A simple CUDA kernel to print the contents of the receive buffer on each GPU
 *
 * Requirements:
 *  - CUDA Toolkit
 *  - NCCL library
 *  - 4 GPUs available on the same node
 *
 * Compile:
 * nvcc -o ncclAllGather.x ncclAllGather.cu -lnccl -Wno-deprecated-gpu-targets
 * 
 * Run:
 * ./ncclAllGather.x    (or submit via Slurm with appropriate GPU request)
 */
#include <stdio.h>
#include "hip/hip_runtime.h"
#include "nccl.h"

__global__ void Dev_print(float *x) {
   
   int i = threadIdx.x;
  
   printf("%1.2f\t", x[i]); 
  
}/*Dev_print*/   


void print_vector(float *in, int n){

 for(int i=0; i < n; i++)
  if(in[i])
   printf("%1.2f\t", in[i]);

}/*print_vector*/


int main(int argc, char* argv[]){

 /*Variables*/
  int size      = 4;
  int nGPUs     = 4;
  int sendcount = 1;
  int DeviceList[4] = {0, 1, 2, 3}; /* (GPUs Id) Testbed on environment with 4 GPUs*/
  
 /*Initializing NCCL with Multiples Devices per Thread*/
  ncclComm_t* comms = (ncclComm_t*)  malloc(sizeof(ncclComm_t)  * nGPUs);  
  hipStream_t* s   = (hipStream_t*)malloc(sizeof(hipStream_t)* nGPUs);
  ncclCommInitAll(comms, nGPUs, DeviceList);

  /*Allocating and initializing device buffers*/
  float** sendbuff = (float**) malloc(nGPUs * sizeof(float*));
  float** recvbuff = (float**) malloc(nGPUs * sizeof(float*));

  /*Host vectors*/ 
  float host_x0[4] = { 10,   0,  0,  0};
  float host_x1[4] = {  0,  20,  0,  0};
  float host_x2[4] = {  0,   0, 30,  0};
  float host_x3[4] = {  0,   0,  0,  40};
    
  print_vector(host_x0, size); 
  print_vector(host_x1, size);
  print_vector(host_x2, size);
  print_vector(host_x3, size);

  for (int i = 0; i < nGPUs; ++i) {

   hipSetDevice(i);

   hipMalloc(&sendbuff[i],  size * sizeof(float));
   hipMalloc(&recvbuff[i],  size * sizeof(float));

    switch(i) { /*Copy from host to devices*/
      case 0 : hipMemcpy(sendbuff[i] , host_x0,   size * sizeof(float), hipMemcpyHostToDevice); break; 
      case 1 : hipMemcpy(sendbuff[i] , host_x1,   size * sizeof(float), hipMemcpyHostToDevice); break; 
      case 2 : hipMemcpy(sendbuff[i] , host_x2,   size * sizeof(float), hipMemcpyHostToDevice); break; 
      case 3 : hipMemcpy(sendbuff[i] , host_x3,   size * sizeof(float), hipMemcpyHostToDevice); break; 
    }

   hipStreamCreate(s+i);

  } 

  ncclGroupStart();
        
        for(int g = 0; g < nGPUs; g++) {
   	      hipSetDevice(g);
          ncclAllGather(sendbuff[g] + g, recvbuff[g], sendcount, ncclFloat, comms[g], s[g]); /*All Gathering the data on GPUs*/
        }

  ncclGroupEnd();


  for(int g = 0; g < nGPUs; g++) {
    hipSetDevice(g); 
    printf("\nThis is device %d\n", g);
    Dev_print <<< 1, size >>> (recvbuff[g]); /*Call the CUDA Kernel: Print vector on GPUs*/
    hipDeviceSynchronize();    
  }

  printf("\n");

  for (int i = 0; i < nGPUs; ++i) { /*Synchronizing CUDA Streams*/
   hipSetDevice(i);
   hipStreamSynchronize(s[i]);
  }

  for (int i = 0; i < nGPUs; ++i) { /*Destroy CUDA Streams*/
   hipSetDevice(i);
   hipFree(sendbuff[i]);
   hipFree(recvbuff[i]);
  }

  for(int i = 0; i < nGPUs; ++i)   /*Finalizing NCCL*/
    ncclCommDestroy(comms[i]);

 /*Freeing memory*/
  hipFree(sendbuff);
  hipFree(recvbuff);

  return 0;

}/*main*/
