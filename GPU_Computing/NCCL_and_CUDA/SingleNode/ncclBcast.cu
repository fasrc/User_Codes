/*
 * NCCL Broadcast (Bcast) Example on a Single Node with Multiple GPUs
 *
 * Description:
 * This program demonstrates how to use NVIDIA's NCCL library with CUDA
 * to broadcast data from one GPU (rank 0) to all other GPUs on a single node.
 * 
 * - GPU 0 initializes a vector with random integers.
 * - The data is broadcast from GPU 0 to all other GPUs using `ncclBcast`.
 * - Each GPU then runs a CUDA kernel to multiply each element of the received vector by 2.
 * - The updated values are printed directly from each GPU.
 *
 * Components:
 *  - CUDA: for device memory allocation, memory copies, kernel launch, and synchronization
 *  - NCCL: for collective communication (Broadcast from GPU 0 to all others)
 *  - CUDA kernel: to modify and print the vector contents from each GPU
 *
 * Requirements:
 *  - CUDA Toolkit
 *  - NCCL library
 *  - At least 2 GPUs on the same node (tested with 4)
 *
 * Compile:
 * nvcc -o ncclBcast.x ncclBcast.cu -lnccl -Wno-deprecated-gpu-targets
 *
 * Run:
 * ./ncclBcast.x    (or submit via Slurm with --gres=gpu:<n>)
 */

#include <hip/hip_runtime.h>
#include <nccl.h>
#include <cstdio>
#include <cstdlib>
 
__global__ void kernel(int *a) 
{
  int index = threadIdx.x;

  a[index] *= 2;
  printf("%d\t", a[index]);

}/*kernel*/
 

void print_vector(int *in, int n){

 for(int i=0; i < n; i++)
  printf("%d\t", in[i]);

 printf("\n");

}/*print_vector*/


int main(int argc, char* argv[]) {

  int data_size = 8 ;
  int nGPUs = 0;
  hipGetDeviceCount(&nGPUs);
  
  int *DeviceList = (int *) malloc (nGPUs     * sizeof(int));
  int *data       = (int*)  malloc (data_size * sizeof(int));
  int **d_data    = (int**) malloc (nGPUs     * sizeof(int*));
  
  for(int i = 0; i < nGPUs; i++)
      DeviceList[i] = i;
  
  /*Initializing NCCL with Multiples Devices per Thread*/
  ncclComm_t* comms = (ncclComm_t*)  malloc(sizeof(ncclComm_t)  * nGPUs);  
  hipStream_t* s   = (hipStream_t*)malloc(sizeof(hipStream_t)* nGPUs);
  ncclCommInitAll(comms, nGPUs, DeviceList);
  
  /*Population the data vector*/
  for(int i = 0; i < data_size; i++)
      data[i] = rand()%(10-2)*2;
 
  print_vector(data, data_size);
      
  for(int g = 0; g < nGPUs; g++) {
      hipSetDevice(DeviceList[g]);
      hipStreamCreate(&s[g]);
      hipMalloc(&d_data[g], data_size * sizeof(int));
     
      if(g == 0)  /*Copy from Host to Device*/
         hipMemcpy(d_data[g], data, data_size * sizeof(int), hipMemcpyHostToDevice);
  }
        
  ncclGroupStart();
 
  		for(int g = 0; g < nGPUs; g++) {
  	  	    hipSetDevice(DeviceList[g]);
    	  	    ncclBcast(d_data[g], data_size, ncclInt, 0, comms[g], s[g]); /*Broadcasting it to all*/
  		}

  ncclGroupEnd();       

  for (int g = 0; g < nGPUs; g++) {
      hipSetDevice(DeviceList[g]);
      printf("\nThis is device %d\n", g);
      kernel <<< 1 , data_size >>> (d_data[g]);/*Call the CUDA Kernel: The code multiple the vector position per 2 on GPUs*/
      hipDeviceSynchronize();             
  }

  printf("\n");

  for (int g = 0; g < nGPUs; g++) { /*Synchronizing CUDA Streams*/
      hipSetDevice(DeviceList[g]);
      hipStreamSynchronize(s[g]);
  }
 
  for(int g = 0; g < nGPUs; g++) {  /*Destroy CUDA Streams*/
      hipSetDevice(DeviceList[g]);
      hipStreamDestroy(s[g]);
  }

  for(int g = 0; g < nGPUs; g++)    /*Finalizing NCCL*/
     ncclCommDestroy(comms[g]);
  
  /*Freeing memory*/
  free(s);
  free(data); 
  free(DeviceList);

  hipFree(d_data);

  return 0;

}/*main*/

