/*
 * NCCL Reduce Example on a Single Node with Multiple GPUs
 *
 * Description:
 * This program demonstrates how to use NVIDIA's NCCL library with CUDA
 * to perform a Reduce operation on vectors distributed across multiple GPUs.
 * 
 * - Each GPU starts with identical input vectors x = [1, 1, ..., 1] and y = [2, 2, ..., 2].
 * - NCCL's `ncclReduce` (with `ncclSum` operation) is used to sum the vectors
 *   from all GPUs and send the result to GPU 0.
 * - A CUDA kernel on each GPU then performs a simple dot product-like operation
 *   using the reduced vectors.
 * - The partial dot product is printed directly from the device.
 *
 * Components:
 *  - CUDA: for device memory management, kernel execution, and synchronization
 *  - NCCL: for collective communication (Reduce with sum operation to GPU 0)
 *  - CUDA kernel: to compute and print a simplified dot product from each GPU
 *
 * Requirements:
 *  - CUDA Toolkit
 *  - NCCL library
 *  - At least 2 GPUs on the same node (tested with 4)
 *
 * Compile:
 * nvcc -o ncclReduce.x ncclReduce.cu -lnccl -Wno-deprecated-gpu-targets
 *
 * Run:
 * ./ncclReduce.x    (or submit via Slurm with --gres=gpu:<n>)
 */

#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <string>
#include <vector>
#include <nccl.h>

__global__ void Dev_dot(double *x, double *y, int n) {
   
   __shared__ double tmp[512];

   int i = threadIdx.x;
   int t = blockDim.x * blockIdx.x + threadIdx.x;
   
   if (t < n) 
    tmp[i] = x[t];
   
   __syncthreads();

   for (int stride = blockDim.x / 2; stride >  0; stride /= 2) {

      if (i < stride)
         tmp[i] += tmp[i + stride];

      __syncthreads();

   }

   if (threadIdx.x == 0) {
      y[blockIdx.x] = tmp[0];
      printf("\tdot(x,y) = %1.2f\n", y[blockIdx.x]); 
   }

}/*Dev_dot*/     


__global__ void Dev_print(double *x) {
   
   int i = threadIdx.x;
    
   printf("%1.2f\t", x[i]);
   
}/*Dev_print*/     


void print_vector(double *in, int n){

 for(int i=0; i < n; i++)
  printf("%1.2f\t", in[i]);

 printf("\n");

}/*print_vector*/


int main(int argc, char* argv[]) {

  /*Variables*/
  int nGPUs = 0;
  hipGetDeviceCount(&nGPUs);
  int *DeviceList = (int *) malloc ( nGPUs * sizeof(int));

  int data_size = 8;

  double *x          = (double*)    malloc(data_size * sizeof(double));
  double *y          = (double*)    malloc(data_size * sizeof(double)); 
  double **x_d_data  = (double**)   malloc(nGPUs     * sizeof(double*));
  double **y_d_data  = (double**)   malloc(nGPUs     * sizeof(double*));
  double **Sx_d_data = (double**)   malloc(nGPUs     * sizeof(double*));
  double **Sy_d_data = (double**)   malloc(nGPUs     * sizeof(double*));
 
  for (int i = 0; i < nGPUs; ++i)
      DeviceList[i] = i;
  
  /*Initializing NCCL with Multiples Devices per Thread*/
  ncclComm_t* comms = (ncclComm_t*)  malloc(sizeof(ncclComm_t)  * nGPUs);  
  hipStream_t* s   = (hipStream_t*)malloc(sizeof(hipStream_t)* nGPUs);
  ncclCommInitAll(comms, nGPUs, DeviceList);
      
  /*Population vectors*/
  for(int i = 0; i < data_size; i++){ 
      x[i] = 1;                
      y[i] = 2;
  }                
      
  print_vector(x, data_size); 
  print_vector(y, data_size);
    

  for(int g = 0; g < nGPUs; g++) {
      hipSetDevice(DeviceList[g]);
      hipStreamCreate(&s[g]);

      hipMalloc(&x_d_data[g],    data_size * sizeof(double));
      hipMalloc(&y_d_data[g],    data_size * sizeof(double));
      
      hipMalloc(&Sx_d_data[g],   data_size * sizeof(double));
      hipMalloc(&Sy_d_data[g],   data_size * sizeof(double));
     
      hipMemcpy(x_d_data[g],  x, data_size * sizeof(double), hipMemcpyHostToDevice); /*Copy from Host to Devices*/
      hipMemcpy(y_d_data[g],  y, data_size * sizeof(double), hipMemcpyHostToDevice);       
    }
      
  ncclGroupStart(); 
  
  	for(int g = 0; g < nGPUs; g++) {
   	  hipSetDevice(DeviceList[g]);
          ncclReduce(x_d_data[g], Sx_d_data[g], data_size, ncclDouble, ncclSum, 0, comms[g], s[g]); /*Reducing x vector*/
          ncclReduce(y_d_data[g], Sy_d_data[g], data_size, ncclDouble, ncclSum, 0, comms[g], s[g]); /*Reducing y vector*/
        }

  ncclGroupEnd(); 


  for(int g = 0; g < nGPUs; g++) {
      hipSetDevice(DeviceList[g]);            
      printf("\n This is device %d\n", g);
      Dev_dot <<< 1, data_size >>> (Sy_d_data[g], Sx_d_data[g], data_size); /*Call the CUDA Kernel: dot product*/
      hipDeviceSynchronize();  
  }
  
  for (int g = 0; g < nGPUs; g++) { /*Synchronizing CUDA Streams*/
      hipSetDevice(DeviceList[g]);
      hipStreamSynchronize(s[g]);
  }
  
  for(int g = 0; g < nGPUs; g++) { /*Destroy CUDA Streams*/
      hipSetDevice(DeviceList[g]);
      hipStreamDestroy(s[g]);
  }

  for(int g = 0; g < nGPUs; g++) /*Finalizing NCCL*/
     ncclCommDestroy(comms[g]);
  
  /*Freeing memory*/
  free(s);
  free(x);
  free(y);
  free(DeviceList);

  hipFree(x_d_data);
  hipFree(y_d_data);
  hipFree(Sx_d_data);
  hipFree(Sy_d_data);

  return 0;

}/*main*/
