// Build: nvcc -O3 -std=c++17 -o ncclAllGather_nompi.x ncclAllGather_nompi.cu -lnccl
#include <stdio.h>
#include <vector>
#include <string>
#include <unistd.h>
#include <sys/stat.h>
#include <hip/hip_runtime.h>
#include <nccl.h>

#define CHECK_CUDA(cmd) do {                                   \
  hipError_t e = (cmd);                                       \
  if (e != hipSuccess) {                                      \
    fprintf(stderr, "CUDA error %s:%d: %s\n",                  \
            __FILE__, __LINE__, hipGetErrorString(e));        \
    exit(1);                                                   \
  }                                                            \
} while(0)

#define CHECK_NCCL(cmd) do {                                   \
  ncclResult_t r = (cmd);                                      \
  if (r != ncclSuccess) {                                      \
    fprintf(stderr, "NCCL error %s:%d: %s\n",                  \
            __FILE__, __LINE__, ncclGetErrorString(r));        \
    exit(1);                                                   \
  }                                                            \
} while(0)

static int get_env_int(const char* k, int defv) {
  const char* v = getenv(k);
  return v ? atoi(v) : defv;
}

static void print_line_host(const float* a, int n) {
  for (int i = 0; i < n; ++i) printf("%1.2f\t", a[i]);
  printf("\n");
}

// --- Minimal shared-file OOB for ncclUniqueId ---
static void write_unique_id(const char* path, const ncclUniqueId& id){
  FILE* f = fopen(path, "wb");
  if (!f) { perror("fopen write_unique_id"); exit(1); }
  size_t n = fwrite(&id, 1, sizeof(id), f);
  fclose(f);
  if (n != sizeof(id)) { fprintf(stderr, "short write of unique id\n"); exit(1); }
}

static void read_unique_id_blocking(const char* path, ncclUniqueId& id){
  for (int i=0; i<300; ++i) { // wait up to ~30s
    struct stat st;
    if (stat(path, &st) == 0 && st.st_size == (off_t)sizeof(id)) {
      FILE* f = fopen(path, "rb");
      if (!f) { perror("fopen read_unique_id"); exit(1); }
      size_t n = fread(&id, 1, sizeof(id), f);
      fclose(f);
      if (n != sizeof(id)) { fprintf(stderr, "short read of unique id\n"); exit(1); }
      return;
    }
    usleep(100000); // 100 ms
  }
  fprintf(stderr, "Timed out waiting for unique id file: %s\n", path);
  exit(1);
}

// Simple NCCL-based barrier: allreduce a single int and sync stream
static void nccl_barrier(ncclComm_t comm, hipStream_t stream, int* d_token){
  CHECK_NCCL(ncclAllReduce(d_token, d_token, 1, ncclInt, ncclSum, comm, stream));
  CHECK_CUDA(hipStreamSynchronize(stream));
}

__global__ void Dev_print(const float* x, int n) {
  int i = threadIdx.x;
  if (i < n) printf("%1.2f\t", x[i]);
}

int main() {
  // --- Rank/size from Slurm (fallback to torchrun-style envs) ---
  int world_size = get_env_int("SLURM_NTASKS", get_env_int("WORLD_SIZE", 1));
  int world_rank = get_env_int("SLURM_PROCID", get_env_int("RANK", 0));
  int local_rank = get_env_int("SLURM_LOCALID", get_env_int("LOCAL_RANK", 0));

  if (world_size < 1) { fprintf(stderr, "Invalid world_size\n"); return 1; }

  // --- Select device by local rank ---
  int ndev = 0;
  CHECK_CUDA(hipGetDeviceCount(&ndev));
  if (ndev < 1) { fprintf(stderr, "No CUDA devices found.\n"); return 1; }
  int device = local_rank % ndev;
  CHECK_CUDA(hipSetDevice(device));

  // --- Bootstrap NCCL unique id via shared file ---
  const char* jobid = getenv("SLURM_JOB_ID");
  const char* dir   = getenv("SLURM_SUBMIT_DIR");
  if (!dir) dir = getenv("HOME");
  std::string uid_path = std::string(dir ? dir : ".") + "/nccl_uid." + (jobid ? jobid : "default");

  ncclUniqueId id;
  if (world_rank == 0) {
    CHECK_NCCL(ncclGetUniqueId(&id));
    write_unique_id(uid_path.c_str(), id);
  } else {
    read_unique_id_blocking(uid_path.c_str(), id);
  }

  // --- Create NCCL communicator ---
  ncclComm_t comm;
  CHECK_NCCL(ncclCommInitRank(&comm, world_size, id, world_rank));

  // --- Stream & a tiny device token for barriers ---
  hipStream_t stream;
  CHECK_CUDA(hipStreamCreate(&stream));
  int one = 1;
  int* d_token = nullptr;
  CHECK_CUDA(hipMalloc(&d_token, sizeof(int)));
  CHECK_CUDA(hipMemcpy(d_token, &one, sizeof(int), hipMemcpyHostToDevice));

  // --- Problem setup ---
  const int size = world_size;  // gathered length
  const int sendcount = 1;      // each rank contributes one float

  // Host "send" buffer: only position [rank] is nonzero = 10*(rank+1)
  std::vector<float> hsend(size, 0.0f);
  hsend[world_rank] = 10.0f * (world_rank + 1);

  // Device buffers
  float *sendbuff_d = nullptr, *recvbuff_d = nullptr;
  CHECK_CUDA(hipMalloc(&sendbuff_d, size * sizeof(float)));
  CHECK_CUDA(hipMalloc(&recvbuff_d, size * sizeof(float)));
  CHECK_CUDA(hipMemcpyAsync(sendbuff_d, hsend.data(),
                             size * sizeof(float), hipMemcpyHostToDevice, stream));
  CHECK_CUDA(hipStreamSynchronize(stream));

  // Print initial seed values once (rank 0), e.g. 10..(10*world_size)
  if (world_rank == 0) {
    std::vector<float> init(size);
    for (int i = 0; i < size; ++i) init[i] = 10.0f * (i + 1);
    print_line_host(init.data(), size);
  }

  // --- AllGather across nodes using NCCL ---
  CHECK_NCCL(ncclAllGather(/*send*/ sendbuff_d + world_rank,
                           /*recv*/ recvbuff_d,
                           sendcount, ncclFloat, comm, stream));
  CHECK_CUDA(hipStreamSynchronize(stream));

  // --- Ordered printing using NCCL barrier ---
  for (int r = 0; r < world_size; ++r) {
    nccl_barrier(comm, stream, d_token);        // before rank r prints
    if (world_rank == r) {
      printf("\nThis is rank %d, device %d\n", world_rank, device);
      fflush(stdout);
      Dev_print<<<1, size, 0, stream>>>(recvbuff_d, size);
      CHECK_CUDA(hipGetLastError());
      CHECK_CUDA(hipDeviceSynchronize());      // flush device printf
      printf("\n");
      fflush(stdout);
    }
    nccl_barrier(comm, stream, d_token);        // after rank r prints
  }

  // --- Cleanup ---
  CHECK_CUDA(hipFree(sendbuff_d));
  CHECK_CUDA(hipFree(recvbuff_d));
  CHECK_CUDA(hipFree(d_token));
  CHECK_CUDA(hipStreamDestroy(stream));
  ncclCommDestroy(comm);

  return 0;
}
