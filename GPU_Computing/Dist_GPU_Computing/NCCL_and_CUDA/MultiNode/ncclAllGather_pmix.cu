#include "hip/hip_runtime.h"
/*
 * =============================================================================
 *  NCCL + CUDA AllGather (No MPI) with PMIx Bootstrap
 * =============================================================================
 *
 *  Overview
 *  --------
 *  Demonstrates a fully MPI-free, multi-node NCCL program that uses **PMIx**
 *  for out-of-band bootstrap/rendezvous and CUDA for device work. Each process
 *  controls exactly one GPU (chosen by its per-node local rank), contributes a
 *  single float equal to `10*(rank+1)`, and participates in an `ncclAllGather`
 *  so every rank receives the full vector:
 *
 *      [10.0, 20.0, 30.0, ..., 10.0 * world_size]
 *
 *  Bootstrap (via PMIx)
 *  --------------------
 *  - Rank 0 calls `ncclGetUniqueId()` and **publishes** it using:
 *      `PMIx_Put(PMIX_GLOBAL, "nccl_uid", ...)` + `PMIx_Commit()`
 *  - All ranks then enter `PMIx_Fence` with **PMIX_COLLECT_DATA=true** to push
 *    KVs to the server.
 *  - Non-root ranks fetch the UID with `PMIx_Get` **from the publisher proc**
 *    `{ nspace, rank=0 }` (some PMIx stacks require targeting the origin).
 *
 *  What this shows
 *  ---------------
 *  - Multi-node collectives using **NCCL only** (no MPI) for data movement
 *  - **PMIx** for rank/size discovery and exchanging the `ncclUniqueId`
 *  - One-process-per-GPU pattern across nodes
 *  - Ordered, readable output using PMIx fences as barriers
 *
 *  Requirements
 *  ------------
 *  - CUDA Toolkit
 *  - NCCL library
 *  - PMIx runtime
 *  - A cluster with ≥ 1 GPU per process (example assumes 2 nodes × 4 GPUs)
 *
 *  Build
 *  -----
 *  nvcc -O3 -std=c++17 -o ncclAllGather_pmix.x ncclAllGather_pmix.cu -lnccl -lpmix
 *
 *  Expected output (shape)
 *  -----------------------
 *  - Rank 0 prints the seed once:
 *      10.00 20.00 30.00 ... (up to 10*world_size)
 *  - Then, for ranks 0..world_size-1 in order:
 *      This is rank R, device D
 *          10.00 20.00 30.00 ... (same gathered vector, printed from device)
 *
 *  Notes / Troubleshooting
 *  -----------------------
 *  - If `PMIx_Get("nccl_uid")` times out:
 *      • Ensure a **collecting fence** runs after `PMIx_Put` (`PMIX_COLLECT_DATA=true`).
 *      • Query the **publisher proc** `{ nspace, rank=0 }`, not your own proc.
 *      • Launch with the site-supported PMIx flavor (e.g., `--mpi=pmix_v3`/`v5`).
 *  - Device selection: `device = local_rank % cudaDeviceCount()`.
 *  - For sites where Put/Get is restricted, you can swap to `PMIx_Publish` /
 *    `PMIx_Lookup` or use a tiny TCP rendezvous—still MPI-free.
 *
 *  Author: (Your Name)
 *  License: MIT (or your preferred license)
 * =============================================================================
 */

#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <vector>

#include <hip/hip_runtime.h>
#include <nccl.h>
#include <pmix.h>

#define CHECK_CUDA(cmd) do {                                  \
  hipError_t e = (cmd);                                      \
  if (e != hipSuccess) {                                     \
    fprintf(stderr, "CUDA error %s:%d: %s\n",                 \
            __FILE__, __LINE__, hipGetErrorString(e));       \
    exit(1);                                                  \
  }                                                           \
} while (0)

#define CHECK_NCCL(cmd) do {                                  \
  ncclResult_t r = (cmd);                                     \
  if (r != ncclSuccess) {                                     \
    fprintf(stderr, "NCCL error %s:%d: %s\n",                 \
            __FILE__, __LINE__, ncclGetErrorString(r));       \
    exit(1);                                                  \
  }                                                           \
} while (0)

#define CHECK_PMIX(rc, what) do {                               \
  if ((rc) != PMIX_SUCCESS) {                                   \
    fprintf(stderr, "PMIx error %s:%d: %s: %s\n",               \
            __FILE__, __LINE__, (what), PMIx_Error_string(rc)); \
    exit(1);                                                    \
  }                                                             \
} while (0)

static int getenv_int(const char* k, int defv) {
  const char* v = getenv(k);
  return v ? atoi(v) : defv;
}

__global__ void Dev_print(const float* x, int n) {
  int i = threadIdx.x;
  if (i < n) printf("%1.2f\t", x[i]);
}

static void print_line_host(const float* a, int n) {
  for (int i = 0; i < n; ++i) printf("%1.2f\t", a[i]);
  printf("\n");
}

static void pmix_fence_collect_all(const pmix_proc_t& me) {
  pmix_proc_t wild;
  PMIX_PROC_CONSTRUCT(&wild);
  std::strncpy(wild.nspace, me.nspace, PMIX_MAX_NSLEN);
  wild.rank = PMIX_RANK_WILDCARD;

  pmix_info_t* info = nullptr;
  PMIX_INFO_CREATE(info, 1);
  bool collect = true;
  PMIX_INFO_LOAD(&info[0], PMIX_COLLECT_DATA, &collect, PMIX_BOOL);

  pmix_status_t rc = PMIx_Fence(&wild, 1, info, 1);
  PMIX_INFO_FREE(info, 1);
  CHECK_PMIX(rc, "PMIx_Fence(PMIX_COLLECT_DATA=true)");

  PMIX_PROC_DESTRUCT(&wild);
}

static void pmix_barrier_all(const pmix_proc_t& me) {
  pmix_proc_t wild;
  PMIX_PROC_CONSTRUCT(&wild);
  std::strncpy(wild.nspace, me.nspace, PMIX_MAX_NSLEN);
  wild.rank = PMIX_RANK_WILDCARD;
  pmix_status_t rc = PMIx_Fence(&wild, 1, nullptr, 0);
  CHECK_PMIX(rc, "PMIx_Fence");
  PMIX_PROC_DESTRUCT(&wild);
}

// Query a job-scoped uint32 (JOB_SIZE or UNIV_SIZE), using wildcard proc
static bool pmix_get_u32_job(const pmix_proc_t& me, const char* key, uint32_t* out) {
  pmix_proc_t pr;
  PMIX_PROC_CONSTRUCT(&pr);
  std::strncpy(pr.nspace, me.nspace, PMIX_MAX_NSLEN);
  pr.rank = PMIX_RANK_WILDCARD;

  pmix_value_t* val = nullptr;
  pmix_status_t rc = PMIx_Get(&pr, key, nullptr, 0, &val);
  PMIX_PROC_DESTRUCT(&pr);
  if (rc != PMIX_SUCCESS || !val) return false;

  bool ok = false;
  if (val->type == PMIX_UINT32) { *out = val->data.uint32; ok = true; }
  else if (val->type == PMIX_SIZE) { *out = static_cast<uint32_t>(val->data.size); ok = true; }
  PMIX_VALUE_RELEASE(val);
  return ok;
}

int main(int /*argc*/, char** /*argv*/) {
  // --- PMIx init ---
  pmix_proc_t me;
  pmix_status_t prc = PMIx_Init(&me, nullptr, 0);
  CHECK_PMIX(prc, "PMIx_Init");
  int world_rank = static_cast<int>(me.rank);

  // --- world_size (robust) ---
  uint32_t wsize_u32 = 0;
  if (!pmix_get_u32_job(me, PMIX_JOB_SIZE, &wsize_u32)) {
    pmix_get_u32_job(me, PMIX_UNIV_SIZE, &wsize_u32);
  }
  if (wsize_u32 == 0) {
    int envsize = getenv_int("PMI_SIZE", -1);
    if (envsize < 0) envsize = getenv_int("SLURM_NTASKS", -1);
    if (envsize < 0) envsize = getenv_int("WORLD_SIZE", -1);
    if (envsize <= 0) {
      fprintf(stderr, "[%d] Could not determine world size via PMIx or env\n", world_rank);
      PMIx_Finalize(nullptr, 0);
      return 1;
    }
    wsize_u32 = static_cast<uint32_t>(envsize);
  }
  int world_size = static_cast<int>(wsize_u32);

  // --- local_rank (robust) ---
  int local_rank = -1;
  {
    pmix_value_t* val = nullptr;
    prc = PMIx_Get(&me, PMIX_LOCAL_RANK, nullptr, 0, &val);
    if (prc == PMIX_SUCCESS && val) {
      if (val->type == PMIX_UINT16)      local_rank = static_cast<int>(val->data.uint16);
      else if (val->type == PMIX_UINT32) local_rank = static_cast<int>(val->data.uint32);
      PMIX_VALUE_RELEASE(val);
    }
    if (local_rank < 0) {
      local_rank = getenv_int("SLURM_LOCALID", -1);
      if (local_rank < 0) local_rank = getenv_int("PMI_LOCAL_RANK", -1);
      if (local_rank < 0) local_rank = getenv_int("OMPI_COMM_WORLD_LOCAL_RANK", -1);
      if (local_rank < 0) local_rank = getenv_int("LOCAL_RANK", -1);
    }
    if (local_rank < 0) {
      fprintf(stderr, "[%d] Could not determine local_rank via PMIx or env\n", world_rank);
      PMIx_Finalize(nullptr, 0);
      return 1;
    }
  }

  // --- select GPU by local rank ---
  int ndev = 0;
  CHECK_CUDA(hipGetDeviceCount(&ndev));
  if (ndev < 1) {
    if (world_rank == 0) fprintf(stderr, "No CUDA devices found.\n");
    PMIx_Finalize(nullptr, 0);
    return 1;
  }
  int device = local_rank % ndev;
  CHECK_CUDA(hipSetDevice(device));

  // --- rank 0 publishes ncclUniqueId; then Fence(collect=true); non-root GETs from publisher proc ---
  ncclUniqueId id;
  if (world_rank == 0) {
    CHECK_NCCL(ncclGetUniqueId(&id));
    pmix_value_t putv;
    PMIX_VALUE_CONSTRUCT(&putv);
    putv.type = PMIX_BYTE_OBJECT;
    putv.data.bo.bytes = reinterpret_cast<char*>(&id);
    putv.data.bo.size  = sizeof(id);
    prc = PMIx_Put(PMIX_GLOBAL, "nccl_uid", &putv);
    CHECK_PMIX(prc, "PMIx_Put nccl_uid");
    prc = PMIx_Commit();
    CHECK_PMIX(prc, "PMIx_Commit");
  }

  // Push KVs to server and make visible
  pmix_fence_collect_all(me);

  if (world_rank != 0) {
    pmix_proc_t publisher;
    PMIX_PROC_CONSTRUCT(&publisher);
    std::strncpy(publisher.nspace, me.nspace, PMIX_MAX_NSLEN);
    publisher.rank = 0;  // <- query the publisher (rank 0)
    pmix_value_t* got = nullptr;

    // Optional "wait" hint (some stacks ignore it, but harmless)
    pmix_info_t* info = nullptr;
    PMIX_INFO_CREATE(info, 1);
    bool wait = true;
    PMIX_INFO_LOAD(&info[0], PMIX_WAIT, &wait, PMIX_BOOL);

    prc = PMIx_Get(&publisher, "nccl_uid", info, 1, &got);
    PMIX_INFO_FREE(info, 1);
    CHECK_PMIX(prc, "PMIx_Get nccl_uid");

    if (!got || got->type != PMIX_BYTE_OBJECT || got->data.bo.size != sizeof(id)) {
      fprintf(stderr, "[%d] Bad nccl_uid object\n", world_rank);
      if (got) PMIX_VALUE_RELEASE(got);
      PMIx_Finalize(nullptr, 0);
      return 1;
    }
    std::memcpy(&id, got->data.bo.bytes, sizeof(id));
    PMIX_VALUE_RELEASE(got);
    PMIX_PROC_DESTRUCT(&publisher);
  }

  // --- NCCL communicator ---
  ncclComm_t comm;
  CHECK_NCCL(ncclCommInitRank(&comm, world_size, id, world_rank));

  // --- CUDA stream ---
  hipStream_t stream;
  CHECK_CUDA(hipStreamCreate(&stream));

  // --- AllGather setup ---
  const int size = world_size;
  const int sendcount = 1;

  std::vector<float> hsend(size, 0.0f);
  hsend[world_rank] = 10.0f * (world_rank + 1);

  float *sendbuff_d = nullptr, *recvbuff_d = nullptr;
  CHECK_CUDA(hipMalloc(&sendbuff_d, size * sizeof(float)));
  CHECK_CUDA(hipMalloc(&recvbuff_d, size * sizeof(float)));
  CHECK_CUDA(hipMemcpyAsync(sendbuff_d, hsend.data(),
                             size * sizeof(float), hipMemcpyHostToDevice, stream));
  CHECK_CUDA(hipStreamSynchronize(stream));

  if (world_rank == 0) {
    std::vector<float> init(size);
    for (int i = 0; i < size; ++i) init[i] = 10.0f * (i + 1);
    print_line_host(init.data(), size);
  }

  CHECK_NCCL(ncclAllGather(sendbuff_d + world_rank, recvbuff_d,
                           sendcount, ncclFloat, comm, stream));
  CHECK_CUDA(hipStreamSynchronize(stream));

  // Ordered prints via PMIx barriers
  for (int r = 0; r < world_size; ++r) {
    pmix_barrier_all(me);
    if (world_rank == r) {
      printf("\nThis is rank %d, device %d\n", world_rank, device);
      fflush(stdout);
      Dev_print<<<1, size, 0, stream>>>(recvbuff_d, size);
      CHECK_CUDA(hipGetLastError());
      CHECK_CUDA(hipDeviceSynchronize());
      printf("\n");
      fflush(stdout);
    }
    pmix_barrier_all(me);
  }

  // Cleanup
  CHECK_CUDA(hipFree(sendbuff_d));
  CHECK_CUDA(hipFree(recvbuff_d));
  CHECK_CUDA(hipStreamDestroy(stream));
  ncclCommDestroy(comm);

  PMIx_Finalize(nullptr, 0);
  return 0;
}

