#include "hip/hip_runtime.h"
// Compile with:
// nvcc -O3 -std=c++17 -o nccl_reduce_mpi.x nccl_reduce_mpi.cu -lnccl -lmpi

#include <cstdio>
#include <cstdlib>
#include <vector>
#include <string>
#include <nccl.h>
#include <mpi.h>
#include <hip/hip_runtime.h>

#define CHECK_CUDA(cmd) do {                                  \
  hipError_t e = cmd;                                        \
  if (e != hipSuccess) {                                     \
    fprintf(stderr, "CUDA error %s:%d: %s\n",                 \
            __FILE__, __LINE__, hipGetErrorString(e));       \
    MPI_Abort(MPI_COMM_WORLD, 1);                             \
  }                                                           \
} while(0)

#define CHECK_NCCL(cmd) do {                                  \
  ncclResult_t r = cmd;                                       \
  if (r != ncclSuccess) {                                     \
    fprintf(stderr, "NCCL error %s:%d: %s\n",                 \
            __FILE__, __LINE__, ncclGetErrorString(r));       \
    MPI_Abort(MPI_COMM_WORLD, 1);                             \
  }                                                           \
} while(0)

__global__ void Dev_dot(double *x, double *y, int n) {
   __shared__ double tmp[512];
   int i = threadIdx.x;
   int t = blockDim.x * blockIdx.x + threadIdx.x;

   if (t < n) tmp[i] = x[t]; else tmp[i] = 0.0;
   __syncthreads();

   for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
      if (i < stride) tmp[i] += tmp[i + stride];
      __syncthreads();
   }

   if (threadIdx.x == 0) {
      y[blockIdx.x] = tmp[0];
      printf("\tdot(x,y) = %1.2f\n", y[blockIdx.x]);
   }
}

static void print_vector_host(const double* in, int n) {
  for (int i = 0; i < n; ++i) printf("%1.2f\t", in[i]);
  printf("\n");
}

int main(int argc, char** argv) {
  MPI_Init(&argc, &argv);

  int world_rank = -1, world_size = 0;
  MPI_Comm_rank(MPI_COMM_WORLD, &world_rank);
  MPI_Comm_size(MPI_COMM_WORLD, &world_size);

  // Determine local rank per node (for device selection)
  MPI_Comm local_comm;
  MPI_Comm_split_type(MPI_COMM_WORLD, MPI_COMM_TYPE_SHARED, 0, MPI_INFO_NULL, &local_comm);
  int local_rank = -1;
  MPI_Comm_rank(local_comm, &local_rank);

  int ndev = 0;
  CHECK_CUDA(hipGetDeviceCount(&ndev));
  if (ndev < 1) {
    if (world_rank == 0) fprintf(stderr, "No CUDA devices found.\n");
    MPI_Abort(MPI_COMM_WORLD, 1);
  }
  int device = local_rank % ndev;
  CHECK_CUDA(hipSetDevice(device));

  // Bootstrap NCCL with MPI
  ncclUniqueId id;
  if (world_rank == 0) {
    CHECK_NCCL(ncclGetUniqueId(&id));
  }
  MPI_Bcast(&id, sizeof(id), MPI_BYTE, 0, MPI_COMM_WORLD);

  ncclComm_t comm;
  CHECK_NCCL(ncclCommInitRank(&comm, world_size, id, world_rank));

  hipStream_t stream;
  CHECK_CUDA(hipStreamCreate(&stream));

  const int data_size = 8; // same as your original
  std::vector<double> hx(data_size, 1.0), hy(data_size, 2.0);

  double *x_d = nullptr, *y_d = nullptr, *Sx_d = nullptr, *Sy_d = nullptr;
  CHECK_CUDA(hipMalloc(&x_d,  data_size * sizeof(double)));
  CHECK_CUDA(hipMalloc(&y_d,  data_size * sizeof(double)));
  CHECK_CUDA(hipMalloc(&Sx_d, data_size * sizeof(double)));
  CHECK_CUDA(hipMalloc(&Sy_d, data_size * sizeof(double)));

  CHECK_CUDA(hipMemcpyAsync(x_d, hx.data(), data_size * sizeof(double), hipMemcpyHostToDevice, stream));
  CHECK_CUDA(hipMemcpyAsync(y_d, hy.data(), data_size * sizeof(double), hipMemcpyHostToDevice, stream));
  CHECK_CUDA(hipStreamSynchronize(stream));

  // Only once: show the initial vectors (like your original top-of-output)
  if (world_rank == 0) {
    print_vector_host(hx.data(), data_size);
    print_vector_host(hy.data(), data_size);
  }
  MPI_Barrier(MPI_COMM_WORLD);

  // Global reduce to root=0
  CHECK_NCCL(ncclGroupStart());
  CHECK_NCCL(ncclReduce((const void*)x_d, (void*)Sx_d,
                        data_size, ncclDouble, ncclSum, 0, comm, stream));
  CHECK_NCCL(ncclReduce((const void*)y_d, (void*)Sy_d,
                        data_size, ncclDouble, ncclSum, 0, comm, stream));
  CHECK_NCCL(ncclGroupEnd());

  CHECK_CUDA(hipStreamSynchronize(stream));
  MPI_Barrier(MPI_COMM_WORLD);

  // Print blocks in rank order so the output matches your sample style
  for (int r = 0; r < world_size; ++r) {
    if (world_rank == r) {
      // Leading blank line to match your formatting
      printf("\n This is rank %d, device %d\n", world_rank, device);
      fflush(stdout);

      if (world_rank == 0) {
        // Launch kernel only on rank 0; its printf comes from the device
        Dev_dot<<<1, data_size, 0, stream>>>(Sy_d, Sx_d, data_size);
        CHECK_CUDA(hipGetLastError());
        CHECK_CUDA(hipStreamSynchronize(stream));
      } else {
        // For non-root ranks, mimic your "0.00" line from host
        printf("\tdot(x,y) = 0.00\n");
        fflush(stdout);
      }
    }
    MPI_Barrier(MPI_COMM_WORLD);
  }

  // Cleanup
  CHECK_CUDA(hipFree(x_d));
  CHECK_CUDA(hipFree(y_d));
  CHECK_CUDA(hipFree(Sx_d));
  CHECK_CUDA(hipFree(Sy_d));
  CHECK_CUDA(hipStreamDestroy(stream));
  ncclCommDestroy(comm);
  MPI_Comm_free(&local_comm);
  MPI_Finalize();
  return 0;
}

